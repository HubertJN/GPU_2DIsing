#include "hip/hip_runtime.h"
// -*- mode: C -*-

#include <stdio.h>
#include "gpu_tools.h"
#include "mc_gpu.h"

// Boilerplate error checking code borrowed from stackoverflow
void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}

// Initialisation 
int gpuInitDevice(int deviceIndex, int* gpu_nsms){

    int idev, count, max_shmem;    

    hipError_t err;  // hipError_t is a type defined in hip/hip_runtime.h
    
    // Make sure we have a CUDA capable device to work with
    err = hipGetDeviceCount(&count);

    if ( (count==0) || (err!=hipSuccess) ) {
        fprintf(stdout,"No CUDA supported devices are available in this system.\n");
        return -1;
    } else {
        fprintf(stdout,"Found %d CUDA devices in this system\n",count);
    }


    // hipDeviceProp_t is a type of struct which we will
    // populate with information about the available 
    // GPU compute devices.
    hipDeviceProp_t prop;

    // Loop over the available CUDA devices
    for (idev=0;idev<count;idev++) {

        // Call another CUDA helper function to populate prop
        gpuErrchk( hipGetDeviceProperties(&prop,idev) );

        // Print out a member of the prop struct which tells
        // us the name of the CUDA device. Other members of this
        // struct tell us the clock speed and compute capability
        // of the device.
        fprintf(stdout,"Device %d : %s\n",idev,prop.name);
        fprintf(stdout,"================================\n");
        fprintf(stdout,"Number of SMs       : %d\n",prop.multiProcessorCount);
        fprintf(stdout,"Max SHMEM per block : %ld KB\n",prop.sharedMemPerBlock/1024);
        max_shmem = prop.sharedMemPerBlock;

      	int value;
      	gpuErrchk(hipDeviceGetAttribute(&value, hipDeviceAttributeSharedMemPerBlockOptin, idev));
	      fprintf(stdout,"Max Opt in shared   : %d KB\n",value/1024);




	//fprintf(stderr,"Warp size           : %d\n",prop.warpSize);
        //fprintf(stderr,"Global DRAM         : %ld\n",prop.totalGlobalMem);
	fprintf(stdout,"Recommended ngrids  : %d\n", 4*prop.warpSize*prop.multiProcessorCount);
  fprintf(stdout,"\n");
  


  double dvalue = (8.0 * max_shmem) / (32);
  int maxL = (int)sqrt(dvalue);
  maxL = int(sqrt(32*(maxL*maxL/32)));

  fprintf(stdout,"Estimated largest L for shared memory code path : %d\n", maxL);
  fprintf(stdout,"(Assumes threadsPerBlock = 32)\n");
  fprintf(stdout,"\n");

    }


    // Only set device if specified 
    if (deviceIndex != -1) {
        gpuErrchk( hipSetDevice(deviceIndex) );
    }

    gpuErrchk( hipGetDevice(&idev ) );
    fprintf(stdout,"Using CUDA device : %d\n",idev);
    

    // Store these for use elsewhere
    //gpu_idev = idev;
    gpuErrchk( hipGetDeviceProperties(&prop,idev) );
    *gpu_nsms = prop.multiProcessorCount;

    fflush(stdout);

    return idev;
    
}


__global__ void populate_random(int length, float *rnd_array, hiprandState *state){

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < length){

        // 4 random numbers
        float rnd = hiprand_uniform(&state[idx]);

        // use one of these
        rnd_array[idx] = rnd;

    }       

    return;

}



// Kernel to initialise RNG on the GPU. Used the cuRAND device API with one
// RNG sequence per CUDA thread.
__global__ void init_gpurand(unsigned long long seed, int ngrids, hiprandState *state){

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx<ngrids){

        unsigned long long seq = (unsigned long long)idx;
        
        // Seperate subsequence for each thread
        hiprand_init(seed, seq, 0ull, &state[idx]);
    
        // Different seed for each thread (faster but risky)
        //hiprand_init(seed+23498*idx, 0ull, 0ull, &state[idx]);
    }


  }


void gpuInitGrid(int L, int ngrids, int threadsPerBlock, int* ising_grids, int** d_ising_grids, int** d_neighbour_list){

    // Allocate threads to thread blocks
    int blocksPerGrid = ngrids/threadsPerBlock;
    if (ngrids%threadsPerBlock!=0) { blocksPerGrid += 1; }

    // Device copy of Ising grid configurations
    gpuErrchk( hipMalloc(d_ising_grids,L*L*ngrids*sizeof(int)) );

    // Populate from host copy
    gpuErrchk( hipMemcpy(*d_ising_grids,ising_grids,L*L*ngrids*sizeof(int),hipMemcpyHostToDevice) );

    // Neighbours
    gpuErrchk (hipMalloc((void **)d_neighbour_list, L*L*4*sizeof(int)) );
    preComputeNeighbours_gpu(L, *d_ising_grids, *d_neighbour_list);

}


void gpuInitRand(int ngrids, int threadsPerBlock, unsigned long rngseed, hiprandState** d_state){


    // Allocate threads to thread blocks
    int blocksPerGrid = ngrids/threadsPerBlock;
    if (ngrids%threadsPerBlock!=0) { blocksPerGrid += 1; }
  
    // Initialise GPU RNG
    gpuErrchk (hipMalloc((void **)d_state, ngrids*sizeof(hiprandState)) );
    unsigned long long gpuseed = (unsigned long long)rngseed;
    init_gpurand<<<blocksPerGrid,threadsPerBlock>>>(gpuseed, ngrids, *d_state);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Test CUDA RNG (DEBUG)
    
    /*
    float   *testrnd = (float *)malloc(ngrids*sizeof(float));
    float *d_testrnd;
    gpuErrchk( hipMalloc(&d_testrnd, ngrids*sizeof(float)) );

    int trial;
    for (trial=0;trial<10;trial++){

      populate_random<<<blocksPerGrid,threadsPerBlock>>>(ngrids, d_testrnd, *d_state);
      gpuErrchk( hipPeekAtLastError() );
      gpuErrchk( hipDeviceSynchronize() );
      gpuErrchk( hipMemcpy(testrnd, d_testrnd, ngrids*sizeof(float), hipMemcpyDeviceToHost) );

      int i;
      for (i=0;i<ngrids;i++){
        printf("Random number on grid %d : %12.4f\n",i,testrnd[i]);
      }
  
    }

    free(testrnd);
    hipFree(d_testrnd);
    exit(EXIT_SUCCESS);
    */

}

int select_gpu_method(int L, int ngrids, int threadsPerBlock, int gpu_device ) {

  hipDeviceProp_t prop;
  
  /* Max shared memory available to a thread block */
  gpuErrchk( hipGetDeviceProperties(&prop,gpu_device) );
  int max_shmem = prop.sharedMemPerBlock;

  double value = (8.0 * max_shmem) / (threadsPerBlock);
  int maxL = (int)sqrt(value);
  maxL = int(sqrt(32*(maxL*maxL/32)));

  int method;
  if ( L <= maxL ) {
   //printf("Problem size fits into shared memory with multi-spin coding : using method 2.\n");
    method = 2;
  } else {
    //printf("Problem size too large for shared memory. Using (slow) global memory.\n");
    method = 0;
  }
  
  //printf("For reference, estimated largest L for method 2 : %d\n", maxL);

  fflush(stdout);
  
  return method;

}




