#include "hip/hip_runtime.h"
// -*- mode: C -*-

#include <stdio.h>
#include "gpu_tools.h"
#include "mc_gpu.h"

// Boilerplate error checking code borrowed from stackoverflow
void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}

// Initialisation 
int gpuInitDevice(int deviceIndex){

    int idev, count;    

    hipError_t err;  // hipError_t is a type defined in hip/hip_runtime.h
    
    // Make sure we have a CUDA capable device to work with
    err = hipGetDeviceCount(&count);

    if ( (count==0) || (err!=hipSuccess) ) {
        fprintf(stdout,"No CUDA supported devices are available in this system.\n");
        return -1;
    } else {
        fprintf(stdout,"Found %d CUDA devices in this system\n",count);
    }


    // hipDeviceProp_t is a type of struct which we will
    // populate with information about the available 
    // GPU compute devices.
    hipDeviceProp_t prop;

    // Loop over the available CUDA devices
    for (idev=0;idev<count;idev++) {

        // Call another CUDA helper function to populate prop
        gpuErrchk( hipGetDeviceProperties(&prop,idev) );

        // Print out a member of the prop struct which tells
        // us the name of the CUDA device. Other members of this
        // struct tell us the clock speed and compute capability
        // of the device.
        fprintf(stdout,"Device %d : %s\n",idev,prop.name);
        fprintf(stdout,"================================\n");
        fprintf(stdout,"Number of SMs       : %d\n",prop.multiProcessorCount);
        fprintf(stdout,"Max SHMEM per block : %ld KB\n",prop.sharedMemPerBlock/1024);

	int value;
	gpuErrchk(hipDeviceGetAttribute(&value, hipDeviceAttributeSharedMemPerBlockOptin, idev));
	fprintf(stdout,"Max Opt in shared   : %ld KB\n",value/1024);


	//fprintf(stderr,"Warp size           : %d\n",prop.warpSize);
        //fprintf(stderr,"Global DRAM         : %ld\n",prop.totalGlobalMem);
	fprintf(stdout,"Recommended ngrids  : %d\n", 4*prop.warpSize*prop.multiProcessorCount);
	
        fprintf(stdout,"\n");

    }


    // Only set device if specified 
    if (deviceIndex != -1) {
        gpuErrchk( hipSetDevice(deviceIndex) );
    }

    gpuErrchk( hipGetDevice(&idev ) );
    fprintf(stdout,"Using CUDA device : %d\n",idev);
    

    fflush(stdout);

    return idev;
    
}


__global__ void populate_random(int length, float *rnd_array, hiprandState *state){

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < length){

        // 4 random numbers
        float rnd = hiprand_uniform(&state[idx]);

        // use one of these
        rnd_array[idx] = rnd;

    }       

    return;

}



// Kernel to initialise RNG on the GPU. Used the cuRAND device API with one
// RNG sequence per CUDA thread.
__global__ void init_gpurand(unsigned long long seed, int ngrids, hiprandState *state){

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx<ngrids){

        unsigned long long seq = (unsigned long long)idx;
        
        // Seperate subsequence for each thread
        hiprand_init(seed, seq, 0ull, &state[idx]);
    
        // Different seed for each thread (faster but risky)
        //hiprand_init(seed+23498*idx, 0ull, 0ull, &state[idx]);
    }


  }


void gpuInitGrid(int L, int ngrids, int threadsPerBlock, int* ising_grids, int** d_ising_grids, int** d_neighbour_list){

    // Allocate threads to thread blocks
    int blocksPerGrid = ngrids/threadsPerBlock;
    if (ngrids%threadsPerBlock!=0) { blocksPerGrid += 1; }

    // Device copy of Ising grid configurations
    gpuErrchk( hipMalloc(d_ising_grids,L*L*ngrids*sizeof(int)) );

    // Populate from host copy
    gpuErrchk( hipMemcpy(*d_ising_grids,ising_grids,L*L*ngrids*sizeof(int),hipMemcpyHostToDevice) );

    // Neighbours
    gpuErrchk (hipMalloc((void **)d_neighbour_list, L*L*4*sizeof(int)) );
    preComputeNeighbours_gpu(L, *d_ising_grids, *d_neighbour_list);

}


void gpuInitRand(int ngrids, int threadsPerBlock, unsigned long rngseed, hiprandState** d_state){


    // Allocate threads to thread blocks
    int blocksPerGrid = ngrids/threadsPerBlock;
    if (ngrids%threadsPerBlock!=0) { blocksPerGrid += 1; }
  
    // Initialise GPU RNG
    gpuErrchk (hipMalloc((void **)d_state, ngrids*sizeof(hiprandState)) );
    unsigned long long gpuseed = (unsigned long long)rngseed;
    init_gpurand<<<blocksPerGrid,threadsPerBlock>>>(gpuseed, ngrids, *d_state);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Test CUDA RNG (DEBUG)
    
    /*
    float   *testrnd = (float *)malloc(ngrids*sizeof(float));
    float *d_testrnd;
    gpuErrchk( hipMalloc(&d_testrnd, ngrids*sizeof(float)) );

    int trial;
    for (trial=0;trial<10;trial++){

      populate_random<<<blocksPerGrid,threadsPerBlock>>>(ngrids, d_testrnd, *d_state);
      gpuErrchk( hipPeekAtLastError() );
      gpuErrchk( hipDeviceSynchronize() );
      gpuErrchk( hipMemcpy(testrnd, d_testrnd, ngrids*sizeof(float), hipMemcpyDeviceToHost) );

      int i;
      for (i=0;i<ngrids;i++){
        printf("Random number on grid %d : %12.4f\n",i,testrnd[i]);
      }
  
    }

    free(testrnd);
    hipFree(d_testrnd);
    exit(EXIT_SUCCESS);
    */

}

int select_gpu_method(int L, int ngrids, int threadsPerBlock, int gpu_device ) {

  hipDeviceProp_t prop;
  
  /* Max shared memory available to a thread block */
  gpuErrchk( hipGetDeviceProperties(&prop,gpu_device) );
  int max_shmem = 32*1024; prop.sharedMemPerBlock;
  
  /* Number of SMs */
  int num_sms = prop.multiProcessorCount;

  /* Need enough memory for two warps at once
     to overlap compuation and copying */
  double subs = 1.0; //ngrids/(threadsPerBlock*num_sms);

  /* Shared memory required for method 2 */
  int req_shmem = ceil(L*L/8)*threadsPerBlock*subs*sizeof(uint8_t);

  int method;
  if ( req_shmem < max_shmem ) {
    printf("Problem size fits into shared memory with multi-spin coding : using method 2.\n");
    method = 2;
  } else {
    printf("Problem size too large for shared memory. Using (slow) global memory.\n");
    method = 0;
  }


  double value = (8.0 * max_shmem) / (subs*threadsPerBlock);
  int maxL = (int)sqrt(value);
  maxL = int(sqrt(32*(maxL*maxL/32)));
  
  printf("For reference, estimated largest L for method 2 : %d\n", maxL);

  fflush(stdout);
  
  return method;

}




